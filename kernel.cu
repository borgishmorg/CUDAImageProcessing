#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>
#include <algorithm>

using namespace std;
using uchar = unsigned char;

__global__ void kernel(uchar* data, uchar* new_data, unsigned height, unsigned width) {
	float matr[3][3] =
	{
		{0.11111f, 0.11111f, 0.11111f},
		{0.11111f, 0.11111f, 0.11111f},
		{0.11111f, 0.11111f, 0.11111f}
	};
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < height && j < width) {
		float resB = 0.f, resG = 0.f, resR = 0.f;
		for (int di : {-1, 0, 1}) {
			for (int dj : {-1, 0, 1}) {
				int ni = max(0, min(i + di, (int)height - 1));
				int nj = max(0, min(j + dj, (int)width - 1));
				resB += (char)(matr[1 + di][1 + dj] * data[(ni * width + nj) * 3]);
				resG += (char)(matr[1 + di][1 + dj] * data[(ni * width + nj) * 3 + 1]);
				resR += (char)(matr[1 + di][1 + dj] * data[(ni * width + nj) * 3 + 2]);
			}
		}
		new_data[(i * width + j) * 3] = (char) resB;
		new_data[(i * width + j) * 3 + 1] = (char) resG;
		new_data[(i * width + j) * 3 + 2] = (char) resR;
	}
}

int main(){
	ifstream in("picture.bmp", ios::in|ios::binary);
	ofstream out("new_picture.bmp", ofstream::binary);
	
	uchar *picture, *new_picture;

	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&picture, 50*1024*1024, hipHostMallocMapped);
	hipHostAlloc(&new_picture, 50*1024*1024, hipHostMallocMapped);

	int len = 0;

	while (in.read((char*)picture+len, 1)) new_picture[len] = picture[len], len++;
	
	unsigned begin = *(unsigned*)(picture + 10);
	unsigned width = *(unsigned*)(picture + 18);
	unsigned height = *(unsigned*)(picture + 22);

	uchar* data = picture + begin;
	uchar* new_data = new_picture + begin;

	dim3 block(32, 32), numBlock((height+31)/32, (width+31)/32);

	kernel <<<numBlock, block>>> (data, new_data, height, width);

	hipDeviceSynchronize();
	for (int i = 0; i < len; i++)
		out << new_picture[i];
	
    return 0;
}
